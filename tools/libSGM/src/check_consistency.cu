#include "hip/hip_runtime.h"
/*
Copyright 2016 fixstars

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "internal.h"

namespace {
	template<typename SRC_T>
	__global__ void check_consistency_kernel(uint16_t* d_leftDisp, const uint16_t* d_rightDisp, const SRC_T* d_left, int width, int height)  {

		const int j = blockIdx.x * blockDim.x + threadIdx.x;
		const int i = blockIdx.y * blockDim.y + threadIdx.y;

		// left-right consistency check, only on leftDisp, but could be done for rightDisp too

		SRC_T mask = d_left[i * width + j];
		int d = d_leftDisp[i * width + j];
		int k = j - d;
		if (mask == 0 || d <= 0 || (k >= 0 && k < width && abs(d_rightDisp[i * width + k] - d) > 1)) {
			// masked or left-right inconsistent pixel -> invalid
			d_leftDisp[i * width + j] = 0;
		}
	}
}

namespace sgm {
	namespace details {

		void check_consistency(uint16_t* d_left_disp, const uint16_t* d_right_disp, const void* d_src_left, int width, int height, int depth_bits) {

			const dim3 blocks(width / 16, height / 16);
			const dim3 threads(16, 16);
			if (depth_bits == 16) {
				check_consistency_kernel<uint16_t> << < blocks, threads >> > (d_left_disp, d_right_disp, (uint16_t*)d_src_left, width, height);
			}
			else if (depth_bits == 8) {
				check_consistency_kernel<uint8_t> << < blocks, threads >> > (d_left_disp, d_right_disp, (uint8_t*)d_src_left, width, height);
			}
			
			CudaKernelCheck();	
		}

	}
}
