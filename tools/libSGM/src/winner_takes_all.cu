#include "hip/hip_runtime.h"
/*
Copyright 2016 fixstars

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "internal.h"

static const int WTA_PIXEL_IN_BLOCK = 8;

namespace {

	__device__ inline int min_warp(int val) {
		val = min(val, __shfl_xor(val, 16));
		val = min(val, __shfl_xor(val, 8));
		val = min(val, __shfl_xor(val, 4));
		val = min(val, __shfl_xor(val, 2));
		val = min(val, __shfl_xor(val, 1));
		return __shfl(val, 0);
	}

	__global__ void winner_takes_all_kernel64(uint16_t* leftDisp, uint16_t* rightDisp, const uint16_t* __restrict__ d_cost, int width, int height)
	{
		const float uniqueness = 0.95f;
		const int DISP_SIZE = 64;
		int idx = threadIdx.x;
		int x = blockIdx.x * WTA_PIXEL_IN_BLOCK + threadIdx.y;
		int y = blockIdx.y;

		const size_t cost_offset = DISP_SIZE * (y * width + x);
		const uint16_t* current_cost = d_cost + cost_offset;
		__shared__ uint16_t tmp_costs_block[DISP_SIZE * WTA_PIXEL_IN_BLOCK];
		uint16_t* tmp_costs = &tmp_costs_block[DISP_SIZE * threadIdx.y];

		uint32_t tmp_cL1, tmp_cL2;
		uint32_t tmp_cR1, tmp_cR2;

		// right (1)
		tmp_costs[idx] = ((x + idx) >= width) ? 0xffff : *(d_cost + DISP_SIZE * (y * width + (x + idx)) + idx);
		tmp_costs[idx + 32] = ((x + (idx + 32)) >= width) ? 0xffff : *(d_cost + DISP_SIZE * (y * width + (x + idx + 32)) + idx + 32);

		tmp_cL1 = current_cost[idx];
		tmp_cL2 = current_cost[idx + 32];
		tmp_cR1 = tmp_costs[idx];
		tmp_cR2 = tmp_costs[idx + 32];

		tmp_cL1 = (tmp_cL1 << 16) + idx;
		tmp_cL2 = (tmp_cL2 << 16) + idx + 32;
		tmp_cR1 = (tmp_cR1 << 16) + idx;
		tmp_cR2 = (tmp_cR2 << 16) + idx + 32;
		//////////////////////////////////////

		int valL1 = min(tmp_cL1, tmp_cL2);
		int minTempL1 = min_warp(valL1);

		int minCostL1 = (minTempL1 >> 16);
		int minDispL1 = minTempL1 & 0xffff;
		//////////////////////////////////////

		if (idx + x >= width || idx == minDispL1) { tmp_cL1 = 0x7fffffff; }
		if (idx + 32 + x >= width || idx + 32 == minDispL1) { tmp_cL2 = 0x7fffffff; }

		int valL2 = min(tmp_cL1, tmp_cL2);
		int minTempL2 = min_warp(valL2);
		int minCostL2 = (minTempL2 >> 16);
		int minDispL2 = minTempL2 & 0xffff;
		minDispL2 = minDispL2 == 0xffff ? -1 : minDispL2;
		//////////////////////////////////////

		if (idx + x >= width) { tmp_cR1 = 0x7fffffff; }
		if (idx + 32 + x >= width) { tmp_cR2 = 0x7fffffff; }

		int valR1 = min(tmp_cR1, tmp_cR2);
		int minTempR1 = min_warp(valR1);

		int minCostR1 = (minTempR1 >> 16);
		int minDispR1 = minTempR1 & 0xffff;
		if (minDispR1 == 0xffff) { minDispR1 = -1; }

		///////////////////////////////////////////////////////////////////////////////////
		// right (2)
		tmp_costs[idx] = (idx == minDispR1 || (x + idx) >= width) ? 0xffff : tmp_costs[idx];
		tmp_costs[idx + 32] = ((idx + 32) == minDispR1 || (x + (idx + 32)) >= width) ? 0xffff : tmp_costs[idx + 32];

		tmp_cR1 = tmp_costs[idx];
		tmp_cR1 = (tmp_cR1 << 16) + idx;

		tmp_cR2 = tmp_costs[idx + 32];
		tmp_cR2 = (tmp_cR2 << 16) + idx + 32;

		if (idx + x >= width || idx == minDispR1) { tmp_cR1 = 0x7fffffff; }
		if (idx + 32 + x >= width || idx + 32 == minDispR1) { tmp_cR2 = 0x7fffffff; }

		int valR2 = min(tmp_cR1, tmp_cR2); // DS == 64
		int minTempR2 = min_warp(valR2);
		int minCostR2 = (minTempR2 >> 16);
		int minDispR2 = minTempR2 & 0xffff;
		if (minDispR2 == 0xffff) { minDispR2 = -1; }
		///////////////////////////////////////////////////////////////////////////////////

		if (idx == 0) {
			float lhv = minCostL2 * uniqueness;
			leftDisp[y * width + x] = (lhv < minCostL1 && abs(minDispL1 - minDispL2) > 1) ? 0 : minDispL1 + 1; // add "+1" 
			float rhv = minCostR2 * uniqueness;
			rightDisp[y * width + x] = (rhv < minCostR1 && abs(minDispR1 - minDispR2) > 1) ? 0 : minDispR1 + 1; // add "+1" 
		}
	}

	__global__ void winner_takes_all_kernel128(uint16_t* leftDisp, uint16_t* rightDisp, const uint16_t* __restrict__ d_cost, int width, int height)
	{
		const int DISP_SIZE = 128;
		const float uniqueness = 0.95f;

		int idx = threadIdx.x;
		int x = blockIdx.x * WTA_PIXEL_IN_BLOCK + threadIdx.y;
		int y = blockIdx.y;

		const size_t cost_offset = DISP_SIZE * (y * width + x);
		const uint16_t* current_cost = d_cost + cost_offset;
		__shared__ uint16_t tmp_costs_block[DISP_SIZE * WTA_PIXEL_IN_BLOCK];
		uint16_t* tmp_costs = &tmp_costs_block[DISP_SIZE * threadIdx.y];

		uint32_t tmp_cL1, tmp_cL2; uint32_t tmp_cL3, tmp_cL4;
		uint32_t tmp_cR1, tmp_cR2; uint32_t tmp_cR3, tmp_cR4;

		// right (1)
		const int idx_1 = idx * 4 + 0;
		const int idx_2 = idx * 4 + 1;
		const int idx_3 = idx * 4 + 2;
		const int idx_4 = idx * 4 + 3;

		// TODO optimize global memory loads
		tmp_costs[idx_1] = ((x + (idx_1)) >= width) ? 0xffff : d_cost[DISP_SIZE * (y * width + (x + idx_1)) + idx_1]; // d_cost[y][x + idx0][idx0]
		tmp_costs[idx_2] = ((x + (idx_2)) >= width) ? 0xffff : d_cost[DISP_SIZE * (y * width + (x + idx_2)) + idx_2];
		tmp_costs[idx_3] = ((x + (idx_3)) >= width) ? 0xffff : d_cost[DISP_SIZE * (y * width + (x + idx_3)) + idx_3];
		tmp_costs[idx_4] = ((x + (idx_4)) >= width) ? 0xffff : d_cost[DISP_SIZE * (y * width + (x + idx_4)) + idx_4];

		uint2 tmp_vcL1 = *reinterpret_cast<const uint2*>(&current_cost[idx_1]);
		const uint2 idx_v = make_uint2((idx_2 << 16) | idx_1, (idx_4 << 16) | idx_3);

		tmp_cR1 = tmp_costs[idx_1];
		tmp_cR2 = tmp_costs[idx_2];
		tmp_cR3 = tmp_costs[idx_3];
		tmp_cR4 = tmp_costs[idx_4];

		tmp_cL1 = __byte_perm(idx_v.x, tmp_vcL1.x, 0x5410);
		tmp_cL2 = __byte_perm(idx_v.x, tmp_vcL1.x, 0x7632);
		tmp_cL3 = __byte_perm(idx_v.y, tmp_vcL1.y, 0x5410);
		tmp_cL4 = __byte_perm(idx_v.y, tmp_vcL1.y, 0x7632);

		tmp_cR1 = (tmp_cR1 << 16) + idx_1;
		tmp_cR2 = (tmp_cR2 << 16) + idx_2;
		tmp_cR3 = (tmp_cR3 << 16) + idx_3;
		tmp_cR4 = (tmp_cR4 << 16) + idx_4;
		//////////////////////////////////////

		int valL1 = min(min(tmp_cL1, tmp_cL2), min(tmp_cL3, tmp_cL4));
		int minTempL1 = min_warp(valL1);

		int minCostL1 = (minTempL1 >> 16);
		int minDispL1 = minTempL1 & 0xffff;
		//////////////////////////////////////
		if (idx_1 + x >= width || idx_1 == minDispL1) { tmp_cL1 = 0x7fffffff; }
		if (idx_2 + x >= width || idx_2 == minDispL1) { tmp_cL2 = 0x7fffffff; }
		if (idx_3 + x >= width || idx_3 == minDispL1) { tmp_cL3 = 0x7fffffff; }
		if (idx_4 + x >= width || idx_4 == minDispL1) { tmp_cL4 = 0x7fffffff; }

		int valL2 = min(min(tmp_cL1, tmp_cL2), min(tmp_cL3, tmp_cL4));
		int minTempL2 = min_warp(valL2);
		int minCostL2 = (minTempL2 >> 16);
		int minDispL2 = minTempL2 & 0xffff;
		minDispL2 = minDispL2 == 0xffff ? -1 : minDispL2;
		//////////////////////////////////////

		if (idx_1 + x >= width) { tmp_cR1 = 0x7fffffff; }
		if (idx_2 + x >= width) { tmp_cR2 = 0x7fffffff; }
		if (idx_3 + x >= width) { tmp_cR3 = 0x7fffffff; }
		if (idx_4 + x >= width) { tmp_cR4 = 0x7fffffff; }

		int valR1 = min(min(tmp_cR1, tmp_cR2), min(tmp_cR3, tmp_cR4));
		int minTempR1 = min_warp(valR1);

		int minCostR1 = (minTempR1 >> 16);
		int minDispR1 = minTempR1 & 0xffff;
		if (minDispR1 == 0xffff) { minDispR1 = -1; }
		///////////////////////////////////////////////////////////////////////////////////
		// right (2)
		tmp_costs[idx_1] = ((idx_1) == minDispR1 || (x + (idx_1)) >= width) ? 0xffff : tmp_costs[idx_1];
		tmp_costs[idx_2] = ((idx_2) == minDispR1 || (x + (idx_2)) >= width) ? 0xffff : tmp_costs[idx_2];
		tmp_costs[idx_3] = ((idx_3) == minDispR1 || (x + (idx_3)) >= width) ? 0xffff : tmp_costs[idx_3];
		tmp_costs[idx_4] = ((idx_4) == minDispR1 || (x + (idx_4)) >= width) ? 0xffff : tmp_costs[idx_4];

		tmp_cR1 = tmp_costs[idx_1];
		tmp_cR1 = (tmp_cR1 << 16) + idx_1;

		tmp_cR2 = tmp_costs[idx_2];
		tmp_cR2 = (tmp_cR2 << 16) + idx_2;

		tmp_cR3 = tmp_costs[idx_3];
		tmp_cR3 = (tmp_cR3 << 16) + idx_3;

		tmp_cR4 = tmp_costs[idx_4];
		tmp_cR4 = (tmp_cR4 << 16) + idx_4;

		if (idx_1 + x >= width || idx_1 == minDispR1) { tmp_cR1 = 0x7fffffff; }
		if (idx_2 + x >= width || idx_2 == minDispR1) { tmp_cR2 = 0x7fffffff; }
		if (idx_3 + x >= width || idx_3 == minDispR1) { tmp_cR3 = 0x7fffffff; }
		if (idx_4 + x >= width || idx_4 == minDispR1) { tmp_cR4 = 0x7fffffff; }

		int valR2 = min(min(tmp_cR1, tmp_cR2), min(tmp_cR3, tmp_cR4));
		int minTempR2 = min_warp(valR2);
		int minCostR2 = (minTempR2 >> 16);
		int minDispR2 = minTempR2 & 0xffff;
		if (minDispR2 == 0xffff) { minDispR2 = -1; }
		///////////////////////////////////////////////////////////////////////////////////

		if (idx == 0) {
			float lhv = minCostL2 * uniqueness;
			leftDisp[y * width + x] = (lhv < minCostL1 && abs(minDispL1 - minDispL2) > 1) ? 0 : minDispL1 + 1; // add "+1" 
			float rhv = minCostR2 * uniqueness;
			rightDisp[y * width + x] = (rhv < minCostR1 && abs(minDispR1 - minDispR2) > 1) ? 0 : minDispR1 + 1; // add "+1" 
		}
	}

}



namespace sgm {
	namespace details {

		void winner_takes_all(const uint16_t* d_scost, uint16_t* d_left_disp, uint16_t* d_right_disp, int width, int height, int disp_size) {
			if (disp_size == 64) {
				dim3 blocks(width / WTA_PIXEL_IN_BLOCK, height);
				dim3 threads(32, WTA_PIXEL_IN_BLOCK);
				winner_takes_all_kernel64 << < blocks, threads >> > (d_left_disp, d_right_disp, d_scost, width, height);
			}
			else if (disp_size == 128) {
				dim3 blocks(width / WTA_PIXEL_IN_BLOCK, height);
				dim3 threads(32, WTA_PIXEL_IN_BLOCK);
				winner_takes_all_kernel128 << < blocks, threads >> > (d_left_disp, d_right_disp, d_scost, width, height);
			}
		}

	}
}