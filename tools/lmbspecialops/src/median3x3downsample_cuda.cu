#include "hip/hip_runtime.h"
//
//  lmbspecialops - a collection of tensorflow ops
//  Copyright (C) 2017  Benjamin Ummenhofer, Huizhong Zhou
//  
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//  
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//  
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
#define EIGEN_USE_GPU
#include "config.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "helper.h"
#include "cuda_helper.h"

using namespace tensorflow;

namespace median3x3downsample_internal
{
  template <class T>
  __global__ void median3x3downsample_kernel(
      T* out, const T* in,
      int z_size, 
      int out_x_size, int out_y_size, int out_xy_size,
      int in_x_size, int in_y_size, int in_xy_size )
  {
    int z = blockIdx.z*blockDim.z + threadIdx.z;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    if( x >= out_x_size || y >= out_y_size || z >= z_size )
      return;

    T value[9];
    int value_idx = 0;
    for( int dy = -1; dy <= 1; ++dy )
    for( int dx = -1; dx <= 1; ++dx )
    {
      int x_ = min(in_x_size-1,max(0,2*x+dx));
      int y_ = min(in_y_size-1,max(0,2*y+dy));
      value[value_idx++] = in[z*in_xy_size+y_*in_x_size+x_];
    }
    {
      for(int j = 1; j < 9; ++j)
      {
        if( value[0] > value[j] )
        {
          T tmp = value[0];
          value[0] = value[j];
          value[j] = tmp;
        }
      }
      for(int j = 2; j < 9; ++j)
      {
        if( value[1] > value[j] )
        {
          T tmp = value[1];
          value[1] = value[j];
          value[j] = tmp;
        }
      }
      for(int j = 3; j < 9; ++j)
      {
        if( value[2] > value[j] )
        {
          T tmp = value[2];
          value[2] = value[j];
          value[j] = tmp;
        }
      }
      for(int j = 4; j < 9; ++j)
      {
        if( value[3] > value[j] )
        {
          T tmp = value[3];
          value[3] = value[j];
          value[j] = tmp;
        }
      }
      for(int j = 5; j < 9; ++j)
      {
        if( value[4] > value[j] )
        {
          T tmp = value[4];
          value[4] = value[j];
          value[j] = tmp;
        }
      }
    }
    int out_idx = z*out_xy_size + y*out_x_size + x;
    out[out_idx] = value[4];
  }
}
using namespace median3x3downsample_internal;


template <class T>
class Median3x3DownsampleOp_GPU : public OpKernel 
{
public:
  explicit Median3x3DownsampleOp_GPU(OpKernelConstruction* construction)
    :OpKernel(construction)
  { }

  void Compute( OpKernelContext* context ) override 
  {
    const Tensor& input_tensor = context->input(0);
    auto input = input_tensor.flat<T>();
    const TensorShape input_shape(input_tensor.shape());
    const int rank = input_shape.dims();
    TensorShape output_shape(input_tensor.shape());
    {
      int idx = rank-1;
      output_shape.set_dim(idx,divup(output_shape.dim_size(idx),2));
      idx = rank-2;
      output_shape.set_dim(idx,divup(output_shape.dim_size(idx),2));
    }
    Tensor* output_tensor = 0;
    OP_REQUIRES_OK(context, context->allocate_output(0, output_shape, &output_tensor));
    auto output = output_tensor->flat<T>();

    int64_t z_size = 1;
    for( int i = 0; i < rank-2; ++i )
      z_size *= output_shape.dim_size(i);
    
    auto device = context->eigen_gpu_device();
    median3x3downsample_gpu(
        device.stream(),
        output.data(), input.data(), 
        z_size, 
        input_shape.dim_size(rank-2),
        input_shape.dim_size(rank-1)
        );
  }

  void median3x3downsample_gpu( 
      const hipStream_t& stream,
      T* out, const T* in, 
      int z_size, 
      int in_y_size, int in_x_size )
  {
    int out_x_size = divup(in_x_size,2);
    int out_y_size = divup(in_y_size,2);
    int out_xy_size = out_x_size*out_y_size;
    int in_xy_size = in_x_size*in_y_size;
    dim3 block(32,4,1);
    dim3 grid;
    grid.x = divup(out_x_size,block.x);
    grid.y = divup(out_y_size,block.y);
    grid.z = divup(z_size,block.z);

    median3x3downsample_kernel<T><<<grid,block,0,stream>>>(
        out, in, 
        z_size,
        out_x_size, out_y_size, out_xy_size,
        in_x_size, in_y_size, in_xy_size
        );
  }

private:

};


#define REG_KB(type)                                                          \
REGISTER_KERNEL_BUILDER(                                                      \
    Name("Median3x3Downsample")                                               \
    .Device(DEVICE_GPU)                                                       \
    .TypeConstraint<type>("T"),                                               \
    Median3x3DownsampleOp_GPU<type>);                                         
REG_KB(float)
REG_KB(double)
#undef REG_KB

