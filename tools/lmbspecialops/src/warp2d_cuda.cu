#include "hip/hip_runtime.h"
//
//  lmbspecialops - a collection of tensorflow ops
//  Copyright (C) 2017  Benjamin Ummenhofer, Huizhong Zhou
//  
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//  
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//  
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
#define EIGEN_USE_GPU
#include "config.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "helper.h"
#include "cuda_helper.h"
#include "Eigen/Core"

using namespace tensorflow;

namespace warp2d_kernel_internal
{
#define CLAMP 1
#define VALUE 2
  template <class T, bool NORMALIZED, int BORDER_MODE>
  __global__ void warp2d_kernel(
      T* out, const T* in, const T* displacements, const T border_value,
      int x_size, int y_size, int z_size, int w_size)
  {
    int w = blockIdx.z * blockDim.z + threadIdx.z;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if( x >= x_size || y >= y_size || w >= w_size )
      return;

    typedef Eigen::Matrix<T,2,1> Vec2;
    typedef Eigen::Matrix<int,2,1> Vec2i;
    typedef Eigen::Matrix<T,4,1> Vec4;
    const int xy_size = x_size*y_size;
    const int xyz_size = xy_size*z_size;
#define IN(w,z,y,x) in[(w)*xyz_size+(z)*xy_size+(y)*x_size+(x)]
#define OUT(w,z,y,x) out[(w)*xyz_size+(z)*xy_size+(y)*x_size+(x)]
#define VECTOR(w,z,y,x) displacements[(w)*2*xy_size+(z)*xy_size+(y)*x_size+(x)]
    Vec2 p1(x,y);
    Vec2 v(VECTOR(w,0,y,x), VECTOR(w,1,y,x));
    if( NORMALIZED )
    {
      v.x() *= x_size;
      v.y() *= y_size;
    }
    Vec2 p2 = p1+v;
    Vec2i p2i = p2.template cast<int>();
    
    T a = p2.x()-p2i.x();
    T b = p2.y()-p2i.y();
    Vec4 weights( (1-a)*(1-b), a*(1-b), (1-a)*b, a*b );
    Vec4 values;

    if( BORDER_MODE == CLAMP )
    {
      int x0, y0, x1, y1, x2, y2, x3, y3;
      x0 = min(x_size-1,max(0,p2i.x()));
      y0 = min(y_size-1,max(0,p2i.y()));
      x1 = min(x_size-1,max(0,p2i.x()+1));
      y1 = min(y_size-1,max(0,p2i.y()));
      x2 = min(x_size-1,max(0,p2i.x()));
      y2 = min(y_size-1,max(0,p2i.y()+1));
      x3 = min(x_size-1,max(0,p2i.x()+1));
      y3 = min(y_size-1,max(0,p2i.y()+1));
      for( int z = 0; z < z_size; ++z )
      {
        values(0) = IN(w,z,y0,x0);
        values(1) = IN(w,z,y1,x1);
        values(2) = IN(w,z,y2,x2);
        values(3) = IN(w,z,y3,x3);
        OUT(w,z,y,x) = values.dot(weights);
      }
    }
    else
    {
      int x0, y0, x1, y1, x2, y2, x3, y3;
      x0 = p2i.x();
      y0 = p2i.y();
      x1 = p2i.x()+1;
      y1 = p2i.y();
      x2 = p2i.x();
      y2 = p2i.y()+1;
      x3 = p2i.x()+1;
      y3 = p2i.y()+1;
      for( int z = 0; z < z_size; ++z )
      {
        if( x0 >= 0 && x3 > 0 && x3 < x_size && y0 >= 0 && y3 > 0 && y3 < y_size )
        {
          values(0) = IN(w,z,y0,x0);
          values(1) = IN(w,z,y1,x1);
          values(2) = IN(w,z,y2,x2);
          values(3) = IN(w,z,y3,x3);
          OUT(w,z,y,x) = values.dot(weights);
        }
        else
        {
          OUT(w,z,y,x) = border_value;
        }
      }
    }
#undef IN
#undef OUT
#undef VECTOR
  }

} 
using namespace warp2d_kernel_internal;




template <class T>
class Warp2dOp_GPU : public OpKernel 
{
public:
  explicit Warp2dOp_GPU(OpKernelConstruction* construction)
    :OpKernel(construction)
  { 
    OP_REQUIRES_OK(construction, construction->GetAttr("normalized", &normalized));
    float value_tmp;
    OP_REQUIRES_OK(construction, construction->GetAttr("border_value", &value_tmp));
    border_value = value_tmp;
    std::string border_mode_str;
    OP_REQUIRES_OK(construction, construction->GetAttr("border_mode", &border_mode_str));
    if( border_mode_str == "clamp" )
      border_mode = CLAMP;
    else 
      border_mode = VALUE;
  }

  void Compute( OpKernelContext* context ) override 
  {
    const Tensor& input_tensor = context->input(0);
    auto input = input_tensor.flat<T>();
    const TensorShape input_shape(input_tensor.shape());

    const Tensor& displacements_tensor = context->input(1);
    auto displacements = displacements_tensor.flat<T>();
    const TensorShape displacements_shape(displacements_tensor.shape());

    Tensor* output_tensor = 0;
    OP_REQUIRES_OK(context, context->allocate_output(0, input_shape, &output_tensor));
    auto output = output_tensor->flat<T>();

    const int input_rank = input_shape.dims();


    int x_size = input_shape.dim_size(input_rank-1);
    int y_size = input_shape.dim_size(input_rank-2);
    int z_size = 1;
    int w_size = 1;
    if( input_rank >= 3 )
      z_size = input_shape.dim_size(input_rank-3);
    for( int i = 0; i < input_rank-3; ++i )
      w_size *= input_shape.dim_size(i);


    auto device = context->eigen_gpu_device();
    warp2d_gpu( 
        device.stream(),
        output.data(),
        input.data(),
        displacements.data(),
        border_value,
        border_mode,
        normalized,
        x_size, y_size, z_size, w_size );
    
  }

  void warp2d_gpu( 
      const hipStream_t& stream,
      T* out, const T* in, const T* displacements,
      const T border_value, const int border_mode, bool normalized,
      int x_size, int y_size, int z_size, int w_size)
  {

    dim3 block(32,4,1);
    dim3 grid;
    grid.x = divup(x_size,block.x);
    grid.y = divup(y_size,block.y);
    grid.z = divup(w_size,block.z);

    if( normalized )
    {
      if( border_mode == CLAMP )
        warp2d_kernel<T,true,CLAMP><<<grid,block,0,stream>>>(
            out, in, displacements, border_value,
            x_size, y_size, z_size, w_size);
      else
        warp2d_kernel<T,true,VALUE><<<grid,block,0,stream>>>(
            out, in, displacements, border_value,
            x_size, y_size, z_size, w_size);
    }
    else
    {
      if( border_mode == CLAMP )
        warp2d_kernel<T,false,CLAMP><<<grid,block,0,stream>>>(
            out, in, displacements, border_value,
            x_size, y_size, z_size, w_size);
      else
        warp2d_kernel<T,false,VALUE><<<grid,block,0,stream>>>(
            out, in, displacements, border_value,
            x_size, y_size, z_size, w_size);
    }

    CHECK_CUDA_ERROR;
  }



private:
  int border_mode;
  T border_value;
  bool normalized;
};

#define REG_KB(type)                                                          \
REGISTER_KERNEL_BUILDER(                                                      \
    Name("Warp2d")                                                            \
    .Device(DEVICE_GPU)                                                       \
    .TypeConstraint<type>("T"),                                               \
    Warp2dOp_GPU<type>);                                                    
REG_KB(float)
REG_KB(double)
#undef REG_KB

